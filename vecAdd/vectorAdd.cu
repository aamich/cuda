/**
 * cuda programming example
 */

/**
 * Vector addition: C = A + B.
 *
 * This sample is a very basic sample that implements element by element
 * vector addition. It is the same as the sample illustrating Chapter 2
 * of the programming guide with some additions like error checking.
 */


#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <time.h>

using namespace std;

__global__ void
vectorAdd(const float *A, const float *B, float *C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        C[i] = A[i] + B[i];
    }
}

__global__ void warmup()
{
    int i=1;
    int j=2;
    i=i+j;
}

/**
 * Host main routine
 */
int
main(void)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    int numElements = 100000000;
    size_t size = numElements * sizeof(float);
    printf("[Vector addition of %d elements]\n", numElements);

    // Allocate the host input vector A
    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);

    // Verify that allocations succeeded
    if (h_A == nullptr || h_B == nullptr || h_C == nullptr)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize the host input vectors
    // 1...n => 
    for (int i = 0; i < numElements; ++i)
    {
        h_A[i] = (float)i/(float)numElements;
        h_B[i] = 1.f - (float)i/(float)numElements;
    }

    // host computation
    auto clk = clock();
    for(int i = 0; i < numElements; ++i)
        h_C[i] = h_A[i] + h_B[i];

    cout << "CPU time: " << (clock()-clk)/ (float)CLOCKS_PER_SEC << " seconds" << endl;

    //GPU warm-up
    for(int i = 0; i < 4; ++i) warmup<<<1, 256>>>();

    clk = clock();
    // Allocate the device input vector A
    float *d_A = nullptr;
    float *d_B = nullptr;
    float *d_C = nullptr;
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);

    if(d_A == nullptr || d_B == nullptr || d_C == nullptr )
    {
        fprintf(stderr, "Failed to allocate device vector\n");
        exit(EXIT_FAILURE);
    }

    // Copy the host input vectors A and B in host memory to the device input vectors in
    // device memory
    printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);
    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    printf("Copy output data from the CUDA device to the host memory\n");
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    cout << "GPU time: " << (clock()-clk)/ (float)CLOCKS_PER_SEC << " seconds" << endl;

    // Verify that the result vector is correct
    for (int i = 0; i < numElements; ++i)
    {
        if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5)
        {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }

    printf("Test PASSED\n");

    // Free device global memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    printf("Done\n");
    return 0;
}

